#include "hip/hip_runtime.h"
#include "simple_rect.cuh"
#include <glad/glad.h>
#include <iostream>
#include <cuda_gl_interop.h>

SimpleRectRenderer::SimpleRectRenderer()
    : BaseRenderer("shaders/rect_simple.vert", "shaders/rect_simple.frag") {
  initialize();
}

void SimpleRectRenderer::setup_base_mesh() {
  float base_mesh[] = {
      // t1
      0.0f, 0.0f, // bottom left
      1.0f, 0.0f, // bottom right
      1.0f, 1.0f, // top right
      // t2
      1.0f, 1.0f,
      0.0f, 1.0f,
      0.0f, 0.0f,
  };

  glBindBuffer(GL_ARRAY_BUFFER, vbo_base_mesh);
  glBufferData(GL_ARRAY_BUFFER, sizeof(base_mesh), base_mesh, GL_STATIC_DRAW);
}

void SimpleRectRenderer::setup_vertex_attributes() {
  // Base mesh vertices
  glBindBuffer(GL_ARRAY_BUFFER, vbo_base_mesh);

  // x y position
  glVertexAttribPointer(0, 2, GL_FLOAT, false, 2 * sizeof(float), (void *)0);
  glEnableVertexAttribArray(0);

  // Instance data
  glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
  
  // offset (x, y)
  glVertexAttribPointer(1, 2, GL_FLOAT, false, BYTES_PER_RECT, (void *)0);
  glVertexAttribDivisor(1, 1);
  glEnableVertexAttribArray(1);
  
  // size (width, height)
  glVertexAttribPointer(2, 2, GL_FLOAT, false, BYTES_PER_RECT, (void *)(2 * sizeof(float)));
  glVertexAttribDivisor(2, 1);
  glEnableVertexAttribArray(2);
  
  // color (r, g, b, a)
  glVertexAttribPointer(3, 4, GL_FLOAT, false, BYTES_PER_RECT, (void *)(4 * sizeof(float)));
  glVertexAttribDivisor(3, 1);
  glEnableVertexAttribArray(3);
}

void SimpleRectRenderer::render_impl(size_t count) {
  glDrawArraysInstanced(GL_TRIANGLES, 0, 6, count);
}


void SimpleRectRenderer::add_rect(float x, float y, float width, float height, glm::vec4 color) {
  data.emplace_back(x);
  data.emplace_back(y);
  data.emplace_back(width);
  data.emplace_back(height);
  data.emplace_back(color.r);
  data.emplace_back(color.g);
  data.emplace_back(color.b);
  data.emplace_back(color.a);
}

void SimpleRectRenderer::add_rect(float2 pos, float2 size, glm::vec4 color) {
  add_rect(pos.x, pos.y, size.x, size.y, color);
}


