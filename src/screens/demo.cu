#include "hip/hip_runtime.h"
#include "demo.cuh"
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>

// CUDA kernel to generate animated circles in a grid
__global__ void generate_circle_grid(unsigned int* circle_data, int grid_size, float time, float base_x, float base_y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= grid_size * grid_size) return;
    
    int row = idx / grid_size;
    int col = idx % grid_size;
    
    // Calculate position
    float spacing = 80.0f;
    float x = base_x + col * spacing;
    float y = base_y + row * spacing;
    
    // Animate with sine waves
    float wave_x = sinf(time * 2.0f + col * 0.5f) * 15.0f;
    float wave_y = cosf(time * 1.5f + row * 0.3f) * 10.0f;
    x += wave_x;
    y += wave_y;
    
    // Animate radius with sine wave
    float base_radius = 15.0f;
    float radius_mod = sinf(time * 3.0f + (row + col) * 0.4f) * 8.0f;
    float radius = (base_radius + radius_mod) * 2.0f; // CircleRenderer doubles radius internally
    
    // Animate color based on position and time
    float hue = fmodf(time * 0.5f + (row * 0.1f + col * 0.15f), 1.0f);
    
    // Convert HSV to RGB (simplified)
    float r, g, b;
    if (hue < 0.33f) {
        r = 1.0f - hue * 3.0f;
        g = hue * 3.0f;
        b = 0.0f;
    } else if (hue < 0.66f) {
        r = 0.0f;
        g = 1.0f - (hue - 0.33f) * 3.0f;
        b = (hue - 0.33f) * 3.0f;
    } else {
        r = (hue - 0.66f) * 3.0f;
        g = 0.0f;
        b = 1.0f - (hue - 0.66f) * 3.0f;
    }
    
    // Pack data into CircleRenderer format: x, y, radius, color
    int base_idx = idx * 4;
    circle_data[base_idx + 0] = __float_as_uint(x);
    circle_data[base_idx + 1] = __float_as_uint(y);
    circle_data[base_idx + 2] = __float_as_uint(radius);
    
    // Pack RGBA color into single uint (r, g, b, a in bytes)
    unsigned char rc = (unsigned char)(r * 255.0f);
    unsigned char gc = (unsigned char)(g * 255.0f);
    unsigned char bc = (unsigned char)(b * 255.0f);
    unsigned char ac = 255; // Full alpha
    
    unsigned int color = rc | (gc << 8) | (bc << 16) | (ac << 24);
    circle_data[base_idx + 3] = color;
}

DemoScreen::DemoScreen(Game &game) : DefaultScreen(game) {}

void DemoScreen::render() {
  render_start();

  auto &res = game.get_resources();
  
  // CUDA Circle Animation Demo
  if (show_cuda_demo) {
    const int grid_size = 8;
    const int total_circles = grid_size * grid_size;
    
    // Ensure CircleRenderer has enough capacity
    res.circle.ensure_vbo_capacity(total_circles);
    
    // Map CUDA buffer
    void* cuda_ptr = res.circle.cuda_map_buffer();
    if (cuda_ptr) {
      // Launch CUDA kernel to generate animated circles
      dim3 blockSize(256);
      dim3 gridSizeKernel((total_circles + blockSize.x - 1) / blockSize.x);
      
      generate_circle_grid<<<gridSizeKernel, blockSize>>>(
        (unsigned int*)cuda_ptr, 
        grid_size, 
        animation_time,
        200.0f,  // base_x (offset to right side)
        -200.0f  // base_y (offset to top)
      );
      
      // Wait for kernel to complete
      hipDeviceSynchronize();
      
      // Unmap buffer
      res.circle.cuda_unmap_buffer();
      
      // Render the CUDA-generated circles
      res.circle.render(total_circles);
    } else {
      std::cerr << "Failed to map CUDA buffer for circle demo" << std::endl;
    }
  }
  
  // Test CircleRenderer (manual circles)
  if (show_circles) {
    res.circle.add_circle(-200.0f, 200.0f, 50.0f, glm::vec4(1.0f, 0.2f, 0.2f, 1.0f)); // Red circle
    res.circle.add_circle(-100.0f, 200.0f, 30.0f, glm::vec4(0.2f, 1.0f, 0.2f, 1.0f)); // Green circle
    res.circle.add_circle(0.0f, 200.0f, 40.0f, glm::vec4(0.2f, 0.2f, 1.0f, 0.8f));    // Blue circle (semi-transparent)
  }
  
  // Test RectRenderer (rounded rectangles)
  if (show_rects) {
    res.rect.add_rect(-200.0f, 50.0f, 80.0f, 60.0f, 10.0f, glm::vec4(1.0f, 0.5f, 0.0f, 1.0f)); // Orange rounded rect
    res.rect.add_rect(-100.0f, 50.0f, 60.0f, 40.0f, 5.0f, glm::vec4(0.5f, 0.0f, 1.0f, 1.0f));  // Purple rounded rect
    res.rect.add_rect(0.0f, 50.0f, 90.0f, 50.0f, 15.0f, glm::vec4(0.0f, 1.0f, 1.0f, 1.0f));    // Cyan rounded rect
  }
  
  // Test SimpleRectRenderer (basic rectangles)
  if (show_simple_rects) {
    res.simple_rect.add_rect(-200.0f, -100.0f, 70.0f, 50.0f, glm::vec4(1.0f, 1.0f, 0.0f, 1.0f)); // Yellow rect
    res.simple_rect.add_rect(-100.0f, -100.0f, 50.0f, 70.0f, glm::vec4(1.0f, 0.0f, 1.0f, 1.0f)); // Magenta rect
    res.simple_rect.add_rect(0.0f, -100.0f, 80.0f, 40.0f, glm::vec4(0.5f, 0.5f, 0.5f, 1.0f));    // Gray rect
  }
  
  // Test LineRenderer
  if (show_lines) {
    res.line.add_line(-200.0f, -250.0f, -100.0f, -200.0f, 8.0f, glm::vec4(1.0f, 0.3f, 0.3f, 1.0f)); // Red line
    res.line.add_line(-100.0f, -250.0f, 0.0f, -200.0f, 5.0f, glm::vec4(0.3f, 1.0f, 0.3f, 1.0f));    // Green line
    res.line.add_line(0.0f, -250.0f, 100.0f, -200.0f, 12.0f, glm::vec4(0.3f, 0.3f, 1.0f, 1.0f));    // Blue line
    
    // Add gradient line (different colors at endpoints)
    res.line.add_line(150.0f, -250.0f, 250.0f, -150.0f, 6.0f, 10.0f, 
                      glm::vec4(1.0f, 0.0f, 0.0f, 1.0f), glm::vec4(0.0f, 0.0f, 1.0f, 1.0f)); // Red to blue gradient
  }
  
  // Test FontRenderer with labels (render in world space so they pan with camera)
  if (show_labels) {
    res.main_font_world.add_text(-200.0f, 280.0f, 30, "Circles", glm::vec4(1.0f, 1.0f, 1.0f, 1.0f));
    res.main_font_world.add_text(-200.0f, 130.0f, 30, "Rounded Rects", glm::vec4(1.0f, 1.0f, 1.0f, 1.0f));
    res.main_font_world.add_text(-200.0f, -20.0f, 30, "Simple Rects", glm::vec4(1.0f, 1.0f, 1.0f, 1.0f));
    res.main_font_world.add_text(-200.0f, -170.0f, 30, "Lines", glm::vec4(1.0f, 1.0f, 1.0f, 1.0f));
  }
  
  // Add control instructions in HUD
  auto left = hud_vp.get_left();
  auto top = hud_vp.get_top();
  res.main_font.add_text(left + 20.0f, top - 30.0f, 20, "Demo Controls:", glm::vec4(0.8f, 0.8f, 0.8f, 1.0f));
  res.main_font.add_text(left + 20.0f, top - 55.0f, 16, "1: Toggle Circles (" + std::string(show_circles ? "ON" : "OFF") + ")", 
                         glm::vec4(show_circles ? 0.5f : 0.3f, show_circles ? 1.0f : 0.3f, show_circles ? 0.5f : 0.3f, 1.0f));
  res.main_font.add_text(left + 20.0f, top - 75.0f, 16, "2: Toggle Rounded Rects (" + std::string(show_rects ? "ON" : "OFF") + ")", 
                         glm::vec4(show_rects ? 1.0f : 0.3f, show_rects ? 0.7f : 0.3f, show_rects ? 0.3f : 0.3f, 1.0f));
  res.main_font.add_text(left + 20.0f, top - 95.0f, 16, "3: Toggle Simple Rects (" + std::string(show_simple_rects ? "ON" : "OFF") + ")", 
                         glm::vec4(show_simple_rects ? 1.0f : 0.3f, show_simple_rects ? 1.0f : 0.3f, show_simple_rects ? 0.3f : 0.3f, 1.0f));
  res.main_font.add_text(left + 20.0f, top - 115.0f, 16, "4: Toggle Lines (" + std::string(show_lines ? "ON" : "OFF") + ")", 
                         glm::vec4(show_lines ? 0.3f : 0.3f, show_lines ? 0.5f : 0.3f, show_lines ? 1.0f : 0.3f, 1.0f));
  res.main_font.add_text(left + 20.0f, top - 135.0f, 16, "5: Toggle Labels (" + std::string(show_labels ? "ON" : "OFF") + ")", 
                         glm::vec4(0.8f, 0.8f, 0.8f, 1.0f));
  res.main_font.add_text(left + 20.0f, top - 155.0f, 16, "6: Toggle CUDA Demo (" + std::string(show_cuda_demo ? "ON" : "OFF") + ")", 
                         glm::vec4(show_cuda_demo ? 1.0f : 0.3f, show_cuda_demo ? 0.8f : 0.3f, show_cuda_demo ? 0.3f : 0.3f, 1.0f));

  render_end();
}

void DemoScreen::update() {
  // Update animation time
  static auto start_time = std::chrono::high_resolution_clock::now();
  auto current_time = std::chrono::high_resolution_clock::now();
  animation_time = std::chrono::duration<float>(current_time - start_time).count();
}

bool DemoScreen::handle_input(SDL_Event event) {
  // multiplex input
  if (DefaultScreen::handle_input(event)) {
    return true;
  } else if (event.type == SDL_KEYDOWN) {
    switch (event.key.keysym.sym) {
      case SDLK_1:
        show_circles = !show_circles;
        return true;
      case SDLK_2:
        show_rects = !show_rects;
        return true;
      case SDLK_3:
        show_simple_rects = !show_simple_rects;
        return true;
      case SDLK_4:
        show_lines = !show_lines;
        return true;
      case SDLK_5:
        show_labels = !show_labels;
        return true;
      case SDLK_6:
        show_cuda_demo = !show_cuda_demo;
        return true;
      default:
        break;
    }
  }
  return false;
}
