#include "hip/hip_runtime.h"
#include "cuda_test.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_test_kernel() {
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

void runCudaTest() {
  cuda_test_kernel<<<1, 4>>>();
  hipDeviceSynchronize();
}